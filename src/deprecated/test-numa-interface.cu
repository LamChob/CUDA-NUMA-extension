#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <numa-interface.h>
//#include <migration.h>

#include <iostream>
#include <chrono>

#include <numaif.h>
#include <hwloc.h>

using namespace std;
using namespace chrono;

__global__ void doSomethingKernel(float *in, int sz) {
    for(int inx = 0; inx < sz; inx+= 1024) 
        in[inx + threadIdx.x] += 5.0f;
}


int main(int argc, char* argv[]) {
	assert(!(argc < 2));

    int n_devices; 
    hipGetDeviceCount(&n_devices);

    float* data; 
    int size = atoi(argv[1]);

    //cout << "Cuda Devices: " << n_devices << " ";
//    cout << "NUMA Devices: " << _get_num_nodes() << endl;
    //numaGetDeviceCount(&n_devices);
    //cout << "Total Devices: "<< n_devices << endl;



    dim3 block(1024); 
    dim3 grid(1);
    cout << size <<  " " << size*sizeof(float);
    if ( numaMallocManaged((void**)&data, (size_t)size*sizeof(float), hipMemAttachGlobal, 0) != hipSuccess){ 
        std::cout << "Malloc Fail: " << hipGetLastError() << std::endl; 
        return 0;
    } 
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) cout << "ERROR1: " << e <<endl;

    cout << " " <<  get_pos(data);
    numaMemPrefetchAsync(data, size*sizeof(float),0);
    e = hipGetLastError();
    if (e != hipSuccess) cout << "ERROR2: " << e <<endl;
    doSomethingKernel<<<grid, block>>>(data, size);
    hipDeviceSynchronize();
    e = hipGetLastError();
    if (e != hipSuccess) cout << "ERROR3: " << e <<endl;
    //cout << "Result: " << data[3] << endl;

    auto t1 = system_clock::now();
    e = numaMemPrefetchAsync(data, size*sizeof(float), 2);
    if ( e != hipSuccess) {
        cout << "prefetch Fail: " << hipGetLastError() << endl;
    } //D2H
    hipDeviceSynchronize();
    auto t2 = system_clock::now();
    double mt = duration_cast<nanoseconds>(t2-t1).count();
    cout <<";" << (size*sizeof(float))/mt << " " << get_pos(data);
    
    
    t1 = system_clock::now();
    e = numaMemPrefetchAsync(data, size*sizeof(float), 3);
    if ( e != hipSuccess) {
        cout << "prefetch Fail: " << hipGetLastError() << endl;
    } //D2H
    hipDeviceSynchronize();
    t2 = system_clock::now();
    mt = duration_cast<nanoseconds>(t2-t1).count();
    cout <<";" << (size*sizeof(float))/mt << " " << get_pos(data);

    numaMemPrefetchAsync(data, size*sizeof(float),0);
    doSomethingKernel<<<grid, block>>>(data, size);
    hipDeviceSynchronize();
    t1 = system_clock::now();
    e = numaMemPrefetchAsync(data, size*sizeof(float), 3);
    if ( e != hipSuccess) {
        cout << "prefetch Fail: " << hipGetLastError() << endl;
    } //D2H
    hipDeviceSynchronize();
    t2 = system_clock::now();
    mt = duration_cast<nanoseconds>(t2-t1).count();
    cout <<";" << (size*sizeof(float))/mt << " " << get_pos(data) <<  ";" << data[0] << endl;

    numaFree(data);
}
